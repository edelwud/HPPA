#include <kernels/launcher-grayscale.cuh>

#include <filters/kernels/filter-operator.cuh>
#include <utils/error_check.hpp>
#include <utils/helpers/helper_border.hpp>

float launchGrayscale(std::map<int, int> &filter, Loader::Image &image) {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    image.data = addImageBorder(image.data, image.height, image.width);
    image.width += 2;
    image.height += 2;

    short *devSource;
    short *devDest;
    size_t pitch;

    checkCudaErrors(hipMallocPitch(&devSource, &pitch, image.width * sizeof(short), image.height));
    checkCudaErrors(hipMallocPitch(&devDest, &pitch, image.width * sizeof(short), image.height));
    checkCudaErrors(hipMemcpy2D(devSource, pitch,
                                 image.data, image.width * sizeof(short),
                                 image.width * sizeof(short), image.height, hipMemcpyHostToDevice));

    dim3 block(32, 16);
    dim3 grid((image.width + block.x / 2 - 1) / block.x / 2, (image.height + block.y / 2 - 1) / block.y / 2);

    hipEventRecord(start);
    filterOperator<<<grid, block>>>(filter, devSource, devDest, pitch, image.width, image.height);
    hipEventRecord(stop);

    checkCudaErrors(hipMemcpy2D(image.data, image.width * sizeof(short), devSource, pitch,
                                 image.width*sizeof(short), image.height, hipMemcpyDeviceToHost));

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    image.data = removeImageBorder(image.data, image.height, image.width);
    image.width -= 2;
    image.height -= 2;

    checkCudaErrors(hipFree(devSource));
    checkCudaErrors(hipFree(devDest));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    return milliseconds;
}