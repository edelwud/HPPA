#include <kernels/launcher-grayscale.cuh>

#include <utils/error_check.hpp>

void launchGrayscale(Filter *filter, Loader::Image image) {
    short * data;
    size_t pitch;

    checkCudaErrors(hipMallocPitch(&data, &pitch, image.width, image.height));
    checkCudaErrors(hipMemcpy2D(data, pitch, image.data, 0, image.width, image.height, hipMemcpyHostToDevice));

}