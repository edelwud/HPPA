#include <kernels/launcher-rgb.cuh>

#include <filters/kernels/filter-operator-rgb.cuh>
#include <utils/error_check.hpp>
#include <utils/helpers/helper_border_rgb.hpp>

float launchRGB(char *filter, Loader::Image &image) {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    image.width *= 3;

    image.data = addImageBorderRGB(image.data, image.height, image.width);
    image.width += 6;
    image.height += 2;

    unsigned char *devSource;
    unsigned char *devDest;
    char *devFilter;
    size_t pitch;

    checkCudaErrors(hipMalloc(&devFilter, 9));
    checkCudaErrors(hipMemcpy(devFilter, filter, 9, hipMemcpyHostToDevice));

    checkCudaErrors(hipMallocPitch(&devSource, &pitch, image.width * sizeof(unsigned char), image.height));
    checkCudaErrors(hipMallocPitch(&devDest, &pitch, image.width * sizeof(unsigned char), image.height));
    checkCudaErrors(hipMemcpy2D(devSource, pitch,
                                 image.data, image.width * sizeof(unsigned char),
                                 image.width * sizeof(unsigned char), image.height, hipMemcpyHostToDevice));

    dim3 block(32, 16);
    dim3 grid((image.width + block.x / 2 - 1) / block.x / 4, (image.height + block.y / 2 - 1) / block.y);

    hipEventRecord(start);
    filterOperatorRGB<<<grid, block>>>(devFilter, devSource, devDest, pitch, image.width, image.height);
    hipEventRecord(stop);

    checkCudaErrors(hipMemcpy2D(image.data, image.width * sizeof(unsigned char), devDest, pitch,
                                 image.width * sizeof(unsigned char), image.height, hipMemcpyDeviceToHost));

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    image.data = removeImageBorderRGB(image.data, image.height, image.width);
    image.width -= 6;
    image.height -= 2;

    image.width /= 3;

    checkCudaErrors(hipFree(devSource));
    checkCudaErrors(hipFree(devDest));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    return milliseconds;
}