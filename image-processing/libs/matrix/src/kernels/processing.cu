#include "hip/hip_runtime.h"
#include <kernels/processing.cuh>

#include <builtin_types.h>
#include <hiprand/hiprand_kernel.h>

#include <kernels/random.cuh>
#include <matrix.hpp>
#include <utils/error_check.hpp>

#include <iostream>

__global__ void processing(const unsigned char *devSource,
                           unsigned char *devDest, long long rows,
                           long long columns) {
  unsigned long long index = (blockDim.x * blockIdx.x + threadIdx.x) * 4;
  if (index < rows * columns) {
    extern __shared__ unsigned char sharedSource[];
    memcpy(sharedSource + index, devSource + index, 4);
    __syncthreads();

    unsigned long long sourceRow = index / columns;
    unsigned long long destOffset = index / 2 + sourceRow * columns / 2;

    memcpy(devDest + destOffset + columns / 2, sharedSource + index, 2);
    memcpy(devDest + destOffset, sharedSource + index + 2, 2);
  }
}

void launchProcessing(unsigned char *matrix, unsigned char *devSource,
                      unsigned char *devDest, long long rows,
                      long long columns) {
  size_t size = rows * columns;
  hipStream_t copySourceStream, launchStream;
  checkCudaErrors(hipStreamCreate(&copySourceStream));
  checkCudaErrors(hipStreamCreate(&launchStream));
  checkCudaErrors(hipMemcpyAsync(devSource, matrix, size,
                                  hipMemcpyHostToDevice, copySourceStream));

  size_t launchSize = size / 4;
  dim3 threadsPerBlock(32);
  dim3 numBlocks((launchSize + threadsPerBlock.x - 1) / threadsPerBlock.x);
  processing<<<numBlocks, threadsPerBlock, size, launchStream>>>(
      devSource, devDest, rows, columns);

  checkCudaErrors(hipStreamDestroy(launchStream));
  checkCudaErrors(hipStreamDestroy(copySourceStream));
}
