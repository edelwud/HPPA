#include "hip/hip_runtime.h"
#include <kernels/shared.cuh>

__global__ void reserve(unsigned char *devMemory, long long n) {
  extern __shared__ int s[];
  long long t = threadIdx.x;
  long long tr = n - t - 1;
  auto *charData = (unsigned char *)s;
  charData[t] = devMemory[t];
  __syncthreads();
  devMemory[t] = charData[tr];
}