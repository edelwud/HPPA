#include <benchmark.cuh>

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <kernels/processing.cuh>
#include <kernels/random.cuh>
#include <utils/error_check.hpp>

MatrixGPU::MatrixGPU(long long n, long long m) : Matrix(n, m) {}

void MatrixGPU::process() {
  unsigned char *devSource;
  unsigned char *devDest;
  checkCudaErrors(hipMalloc((void **)&devSource, size));
  checkCudaErrors(hipMalloc((void **)&devDest, size));

  launchProcessing(matrix, devSource, devDest, rows, columns);

  columns /= 2;
  rows *= 2;

  checkCudaErrors(hipMemcpy(matrix, devDest, size, hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(devSource));
  checkCudaErrors(hipFree(devDest));
}

void MatrixGPU::initialize() {
  unsigned char *devMemory;
  checkCudaErrors(hipMalloc((void **)&devMemory, size));

  launchRandom(devMemory, rows, columns);

  checkCudaErrors(hipMemcpy(matrix, devMemory, size, hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(devMemory));
}
