#include "hip/hip_runtime.h"
#include <kernels/filter-operator-rgb.cuh>

__global__ void filterOperatorRGB(const char *filter, unsigned char *devSource, unsigned char *devDest, size_t pitch, int width, int height) {
    auto offsetX = (blockDim.x * blockIdx.x + threadIdx.x) * 4 + 1;
    auto offsetY = (blockDim.y * blockIdx.y + threadIdx.y) + 1;


#pragma unroll
    for (int i = 0; i < 4; i++) {
        short result = 0;
#pragma unroll
        for (int l = -1, k = 0; k < 3; l++, k++) {
#pragma unroll
            for (int j = -1, m = 0; m < 3; j++, m++) {
                result += devSource[(offsetY + l) * pitch + offsetX + j + i] * filter[k * 3 + m];
            }
        }

        result /= 9;
        if (result > 255)
            result = 255;

        if (result < 0)
            result = 0;

        result += 100;

        memcpy(devDest + offsetY * pitch + offsetX + i, &result, 1);
    }
}