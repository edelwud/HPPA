#include "hip/hip_runtime.h"
#include <kernels/filter-operator-rgb.cuh>

__device__ unsigned char fixIntensity(int intensity) {
    if (intensity > 255)
        return 255;

    if (intensity < 0)
        return 0;

    return (unsigned char) intensity;
}

__global__ void filterOperatorRGB(const char *filter, unsigned char *devSource, unsigned char *devDest, size_t pitch, int width, int height) {
    auto offsetX = (blockDim.x * blockIdx.x + threadIdx.x) * 3 + 3;
    auto offsetY = (blockDim.y * blockIdx.y + threadIdx.y) + 1;

    auto sharedX = threadIdx.x * 9;
    auto sharedY = threadIdx.y + 1;

    __shared__ unsigned char sharedMemory[16 + 2][(32 + 2) * 9];
    for (int i = -1; i < 2; i++) {
        memcpy(
                sharedMemory[sharedY + i] + sharedX,
                devSource + (offsetY + i) * pitch + offsetX - 3,
                9);
    }

    __syncthreads();

    int r = 0;
    int g = 0;
    int b = 0;
#pragma unroll
    for (int i = 0; i < 3; i++) {
#pragma unroll
        for (int j = 0; j < 3; j++) {
            r += sharedMemory[sharedY + i - 1][sharedX + j * 3] * filter[i * 3 + j] / 9;
            g += sharedMemory[sharedY + i - 1][sharedX + j * 3 + 1] * filter[i * 3 + j] / 9;
            b += sharedMemory[sharedY + i - 1][sharedX + j * 3 + 2] * filter[i * 3 + j] / 9;
        }
    }

    r = fixIntensity(r);
    g = fixIntensity(g);
    b = fixIntensity(b);

    memcpy(devDest + offsetY * pitch + offsetX, &r, 1);
    memcpy(devDest + offsetY * pitch + offsetX + 1, &g, 1);
    memcpy(devDest + offsetY * pitch + offsetX + 2, &b, 1);
}