#include "hip/hip_runtime.h"
#include <kernels/filter-operator.cuh>

#include <iostream>

__global__ void filterOperator(std::map<int, int> &filter, short * devSource, short * devDest, size_t pitch, int width, int height) {
    uint2 offset = make_uint2(
            (blockDim.x * blockIdx.x + threadIdx.x + 1) * sizeof(short),
            (blockDim.y * blockIdx.y + threadIdx.y + 1) * sizeof(short)
    );

    devDest[0] = 1;
}