#include "hip/hip_runtime.h"
#include <kernels/filter-operator.cuh>

#include <iostream>

__global__ void filterOperator(const char *filter, unsigned char *devSource, unsigned char *devDest, size_t pitch, int width, int height) {
    auto offsetX = (blockDim.x * blockIdx.x + threadIdx.x) * 4 + 1;
    auto offsetY = (blockDim.y * blockIdx.y + threadIdx.y) + 1;

    __shared__ char kekw[64];

#pragma unroll
    for (int i = 0; i < 4; i++) {
        short result = 0;
        unsigned char *source = devSource + offsetY * pitch + offsetX + i;
#pragma unroll
        for (int l = -1, k = 0; k < 3; l++, k++) {
#pragma unroll
            for (int j = -1, m = 0; m < 3; j++, m++) {
                result += source[l*pitch+j] * filter[k*3+m];
            }
        }

        result /= 9;

        if (result > 255)
            result = 255;

        memcpy(devDest+offsetY * pitch + offsetX + i, &result, 1);
    }
}