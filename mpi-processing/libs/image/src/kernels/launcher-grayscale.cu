#include <kernels/launcher-grayscale.cuh>

#include <filters/kernels/filter-operator-grayscale.cuh>
#include <utils/error_check.hpp>
#include <utils/helpers/helper_border_grayscale.hpp>

float launchGrayscale(char *filter, Loader::Image &image) {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    image.data = addImageBorderGrayscale(image.data, image.height, image.width);
    image.width += 2;
    image.height += 2;

    unsigned char *devSource;
    unsigned char *devDest;
    char *devFilter;
    size_t pitch;

    checkCudaErrors(hipMalloc(&devFilter, 9));
    checkCudaErrors(hipMemcpy(devFilter, filter, 9, hipMemcpyHostToDevice));

    checkCudaErrors(hipMallocPitch(&devSource, &pitch, image.width * sizeof(unsigned char), image.height));
    checkCudaErrors(hipMallocPitch(&devDest, &pitch, image.width * sizeof(unsigned char), image.height));
    checkCudaErrors(hipMemcpy2D(devSource, pitch,
                                 image.data, image.width * sizeof(unsigned char),
                                 image.width * sizeof(unsigned char), image.height, hipMemcpyHostToDevice));

    dim3 block(32, 16);
    dim3 grid((image.width + block.x / 2 - 1) / block.x / 4, (image.height + block.y / 2 - 1) / block.y);

    hipEventRecord(start);
    filterOperatorGrayscale<<<grid, block>>>(devFilter, devSource, devDest, pitch, image.width, image.height);
    hipEventRecord(stop);

    checkCudaErrors(hipMemcpy2D(image.data, image.width * sizeof(unsigned char), devDest, pitch,
                                 image.width * sizeof(unsigned char), image.height, hipMemcpyDeviceToHost));

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    image.data = removeImageBorderGrayscale(image.data, image.height, image.width);
    image.width -= 2;
    image.height -= 2;

    checkCudaErrors(hipFree(devSource));
    checkCudaErrors(hipFree(devDest));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    return milliseconds;
}