#include <matrix-gpu.cuh>

#include <hiprand.h>

#include <kernels/random.cuh>
#include <utils/error_check.hpp>

MatrixGPU::MatrixGPU(int n, int m) : Matrix(n, m) {}

void MatrixGPU::process() {

}

void MatrixGPU::initialize() {
    short* devData;
    checkCudaErrors(hipMalloc((void**)&devData, rows*columns*sizeof(short)));
    random<<<1, rows*columns/2>>>(devData);
    checkCudaErrors(hipMemcpy(matrix, devData, rows * columns * sizeof(short), hipMemcpyDeviceToHost));
}
