#include <matrix-gpu.cuh>

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <kernels/random.cuh>
#include <kernels/processing.cuh>
#include <utils/error_check.hpp>

MatrixGPU::MatrixGPU(int n, int m) : Matrix(n, m) {}

void MatrixGPU::process() {
    launch(devSource, devDest, rows, columns);
    columns /= 2;
    rows *= 2;
    checkCudaErrors(hipMemcpy(matrix, devDest, size, hipMemcpyDeviceToHost));
}

void MatrixGPU::initialize() {
    checkCudaErrors(hipMalloc((void**)&devSource, size));
    checkCudaErrors(hipMalloc((void**)&devDest, size));
}

MatrixGPU::~MatrixGPU() {
    checkCudaErrors(hipFree(devSource));
    checkCudaErrors(hipFree(devDest));
}
