#include "hip/hip_runtime.h"
#include <kernels/processing.cuh>

#include <hiprand/hiprand_kernel.h>
#include <builtin_types.h>

#include <matrix.hpp>
#include <kernels/random.cuh>
#include <utils/error_check.hpp>

#include <iostream>

__global__ void process(const char *devSource, char *devDest, int rows, int columns) {
    unsigned int index = (blockDim.x*blockIdx.x+threadIdx.x)*4;

    if (index >= rows*columns)
        return;

    unsigned int sourceRow = index/columns;
    unsigned int destOffset = index/2+sourceRow*columns/2;

    memcpy(devDest+destOffset, devSource+index+2, 2);
    memcpy(devDest+destOffset+columns/2, devSource+index, 2);
}

void launch(char* devSource, char* devDest, int rows, int columns) {
    size_t size = rows*columns/4;
    dim3 threadsPerBlock(4);
    dim3 numBlocks((size+threadsPerBlock.x-1)/threadsPerBlock.x);

    random<<<numBlocks, threadsPerBlock>>>(devSource);

    char *arr = new char[rows*columns];
    checkCudaErrors(hipMemcpy(arr, devSource, rows*columns, hipMemcpyDeviceToHost));

    std::cout << "RANDOMIZED:" << std::endl;
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < columns; j++) {
            printf("%4d ", arr[i*columns + j]);
        }
        std::cout << std::endl;
    }

    std::cout << "FLEX:" << std::endl;
    process<<<numBlocks, threadsPerBlock>>>(devSource, devDest, rows, columns);
}
