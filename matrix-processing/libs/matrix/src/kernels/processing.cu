#include "hip/hip_runtime.h"
#include <kernels/processing.cuh>

#include <hiprand/hiprand_kernel.h>
#include <builtin_types.h>

#include <matrix.hpp>
#include <kernels/random.cuh>
#include <utils/error_check.hpp>

#include <iostream>

__global__ void processing(const unsigned char *devSource, unsigned char *devDest, int rows, int columns) {
    unsigned int index = (blockDim.x*blockIdx.x+threadIdx.x)*4;

    if (index >= rows*columns)
        return;

    unsigned int sourceRow = index/columns;
    unsigned int destOffset = index/2+sourceRow*columns/2;

    memcpy(devDest+destOffset, devSource+index+2, 2);
    memcpy(devDest+destOffset+columns/2, devSource+index, 2);
}

void launchProcessing(unsigned char* devSource, unsigned char* devDest, int rows, int columns) {
    size_t size = rows*columns/4;
    dim3 threadsPerBlock(4);
    dim3 numBlocks((size+threadsPerBlock.x-1)/threadsPerBlock.x);
    processing<<<numBlocks, threadsPerBlock>>>(devSource, devDest, rows, columns);
}
