#include "hip/hip_runtime.h"
#include <kernels/processing.cuh>

#include <hiprand/hiprand_kernel.h>
#include <builtin_types.h>

#include <matrix.hpp>
#include <kernels/random.cuh>
#include <utils/error_check.hpp>

#include <iostream>

__global__ void processing(const unsigned char *devSource, unsigned char *devDest, long long rows, long long columns) {
    unsigned long long index = (blockDim.x*blockIdx.x+threadIdx.x)*4;

    if (index >= rows*columns)
        return;

    unsigned long long sourceRow = index/columns;
    unsigned long long destOffset = index/2+sourceRow*columns/2;

    memcpy(devDest+destOffset, devSource+index+2, 2);
    memcpy(devDest+destOffset+columns/2, devSource+index, 2);
}

void launchProcessing(unsigned char* devSource, unsigned char* devDest, long long rows, long long columns) {
    size_t size = rows*columns/4;
    dim3 threadsPerBlock(4);
    dim3 numBlocks((size+threadsPerBlock.x-1)/threadsPerBlock.x);
    processing<<<numBlocks, threadsPerBlock>>>(devSource, devDest, rows, columns);
}
