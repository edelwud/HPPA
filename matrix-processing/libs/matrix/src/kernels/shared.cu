#include "hip/hip_runtime.h"
#include <kernels/shared.cuh>

__global__ void reserve(char* devMemory, int n) {
    extern __shared__ int s[];
    int t = threadIdx.x;
    int tr = n-t-1;
    char* charData = (char*)s;
    charData[t] = devMemory[t];
    __syncthreads();
    devMemory[t] = charData[tr];
}