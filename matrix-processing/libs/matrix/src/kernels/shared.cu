#include "hip/hip_runtime.h"
#include <kernels/shared.cuh>

__global__ void reserve(unsigned char* devMemory, int n) {
    extern __shared__ int s[];
    int t = threadIdx.x;
    int tr = n-t-1;
    auto* charData = (unsigned char*)s;
    charData[t] = devMemory[t];
    __syncthreads();
    devMemory[t] = charData[tr];
}