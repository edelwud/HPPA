#include "hip/hip_runtime.h"
#include <kernels/shared.cuh>

__global__ void reserve(short* devMemory, int n) {
    extern __shared__ int s[];
    int t = threadIdx.x;
    int tr = n-t-1;
    short* shortData = (short*)s;
    shortData[t] = devMemory[t];
    __syncthreads();
    devMemory[t] = shortData[tr];
}