#include "hip/hip_runtime.h"
#include <kernels/random.cuh>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void random(unsigned char* devMemory) {
    unsigned int i = (blockDim.x * blockIdx.x + threadIdx.x)*4;
    hiprandState state;
    hiprand_init(clock() + i, 0, 0, &state);

    unsigned int randomGenerated = hiprand(&state);
    memcpy(devMemory+i, &randomGenerated, sizeof(int));
}

void launchRandom(unsigned char* devMemory, int rows, int columns) {
    size_t size = rows*columns/4;
    dim3 threadsPerBlock(4);
    dim3 numBlocks((size+threadsPerBlock.x-1)/threadsPerBlock.x);
    random<<<numBlocks, threadsPerBlock>>>(devMemory);
}