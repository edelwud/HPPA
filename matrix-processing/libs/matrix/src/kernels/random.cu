#include "hip/hip_runtime.h"
#include <kernels/random.cuh>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <utils/error_check.hpp>

__global__ void random(short* devMemory) {
    unsigned int i = (blockDim.x * blockIdx.x + threadIdx.x)*2;
    hiprandState state;
    hiprand_init(clock() + i, 0, 0, &state);

    unsigned int randomGenerated = hiprand(&state);
    memcpy(devMemory+i, &randomGenerated, sizeof(int));
}