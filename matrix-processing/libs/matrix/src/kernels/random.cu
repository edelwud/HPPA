#include "hip/hip_runtime.h"
#include <kernels/random.cuh>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void random(unsigned char* devMemory) {
    unsigned long long i = (blockDim.x * blockIdx.x + threadIdx.x)*4;
    hiprandState state;
    hiprand_init(clock() + i, 0, 0, &state);

    unsigned long long randomGenerated = hiprand(&state);
    memcpy(devMemory+i, &randomGenerated, sizeof(int));
}

void launchRandom(unsigned char* devMemory, long long rows, long long columns) {
    size_t size = rows*columns/4;
    dim3 threadsPerBlock(4);
    dim3 numBlocks((size+threadsPerBlock.x-1)/threadsPerBlock.x);
    random<<<numBlocks, threadsPerBlock>>>(devMemory);
}